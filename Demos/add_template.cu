/*
load the cuda module:
$ module load cuda91/toolkit/9.1.85

to compile: 
$ nvcc -o myadd add_template.cu

to start in interactive session: 
$qsub -I -N add -W group_list=ece569 -q standard -l select=1:ncpus=2:mem=12gb:ngpus=1 -l walltime=00:00:50

to execute: 
$ ./myadd

this template will compile and run the host side as it is. 

exit out of interactive session before running the next 
interactive job or compiling: 
$ exit
*/

#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

// first define the kernel
// later we will add print statement to print thread id and 
// blockid for the 16 blocks and 1 thread/block configuration
// insert your code here 

int main()
{
int i;
int* d_a;
int* d_b;

int* h_a;
int* h_b;

hipEvent_t startEvent, stopEvent;
float elapsedTime;
hipEventCreate(&startEvent);
hipEventCreate(&stopEvent);

int count = 1000;


srand(time(NULL));


h_a = (int*)malloc(count*sizeof(int));
h_b = (int*)malloc(count*sizeof(int));

for (i=0;i<count;i++) {
  h_a[i] = rand()%1000;
  h_b[i] = rand()%1000;
}
printf("before addition\n");
for(i=0;i<5;i++)
   printf("%d and %d\n",h_a[i],h_b[i]);

hipEventRecord(startEvent, 0);

// allocate memory on device, check for failure
// insert your code here 



// copy data to device, check for failure, free device if needed
// insert your code here 


/* 
generic kernel launch: 
b: blocks
t: threads
shmem: amount of shard memory allocated per block, 0 if not defined

AddInts<<<dim3(bx,by,bz), dims(tx,ty,tz),shmem>>>(parameters)
dim3(w,1,1) = dim3(w) = w

AddInts<<<dim3(4,4,2),dim3(8,8)>>>(....)

How many blocks?
How many threads/blocks?
How many threads?

*/

/* 
 1) set the grid size and block size with the dim3 structure and launch the kernel 
 intitially set the block size to 256 and determine the grid size 
 launch the kernel
 
 2) later we will experiment with printing block ids for the configuration of
 16 blocks and 1 thread per block. For this second experiment insert printf statement 
 in the kernel. you will need hipDeviceSynchronize() call after kernel launch to 
 flush the printfs. 
 
*/
// insert your code here 




//if printing from the kernel flush the printfs 
// insert your code here 


// retrieve data from the device, check for error, free device if needed 
// insert your code here 

 
hipEventRecord(stopEvent, 0);
hipEventSynchronize(stopEvent);
hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
printf("Total execution time (ms) %f\n",elapsedTime);
for(i=0;i<5;i++)
   printf("%d \n",h_a[i]);
   
hipEventDestroy(startEvent);
hipEventDestroy(stopEvent);


return 0;
}

