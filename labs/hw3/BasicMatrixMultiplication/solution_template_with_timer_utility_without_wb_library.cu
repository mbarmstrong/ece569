#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
  //@@ Insert code to implement basic matrix multiplication for
  //@@ arbitrary size using global memory. 

  int Row = blockIdx.y * blockDim.y + threadIdx.y;  // calculate row index
  int Col = blockIdx.x * blockDim.x + threadIdx.x;  // calculate column index
  float Cvalue = 0; // accumulated C element value

  // A (m x k) * B (k x n) = C (m x n)
  // # rows in C = # rows in A
  // # columns in C = # columns in B

  if ((Row < numCRows) && (Col < numCColumns)) {
    for (int i = 0; i < numAColumns; i++) 
      Cvalue += A[numAColumns * Row + i] * B[i * numBColumns + Col];
    C[Row * numCColumns + Col] = Cvalue;
  }

}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA; // A matrix on device
  float *deviceB; // B matrix on device
  float *deviceC; // C matrix on device
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  int dim_param;

hipEvent_t astartEvent, astopEvent;
float aelapsedTime;
hipEventCreate(&astartEvent);
hipEventCreate(&astopEvent);

  args = wbArg_read(argc, argv);

 //ali  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows    = numARows;   // set to correct value
  numCColumns = numBColumns;   // set to correct value
  
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));
  
 //ali  wbTime_stop(Generic, "Importing data and creating memory on host");

//ali  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
//ali  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
//ali  wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);
  
 //ali  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here for A, B and C
  
  hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
  hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
  hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));
  
 //ali wbTime_stop(GPU, "Allocating GPU memory.");

 //ali wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here for A and B
  
  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);
  
 //ali wbTime_stop(GPU, "Copying input memory to the GPU.");
  
  // ali: this loop will sweep blocks 4x4, 8x8, 16x16 and 32x32
  dim_param=4;
  for(dim_param=4; dim_param<33; dim_param=dim_param*2) {
    dim3 DimBlock(dim_param,dim_param,1);
    //@@ Initialize the grid dimensions here
    // use dim3 structure for setting grid dimensions
    dim3 DimGrid((numCColumns -1)/dim_param + 1, (numCRows - 1)/dim_param + 1, 1);  // need to fill in DimGrid() call
    //ali  wbTime_start(Compute, "Performing CUDA computation");
    hipEventRecord(astartEvent, 0);
   //@@ Launch the GPU Kernel here
  
  matrixMultiply<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, 
                                      numARows, numAColumns,
                                      numBRows, numBColumns, 
                                      numCRows, numCColumns);

  hipDeviceSynchronize();
  //hipDeviceSynchronize();

  hipEventRecord(astopEvent, 0);
  hipEventSynchronize(astopEvent);
  hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
  printf("Total execution time (ms) %f for block size %d x %d matrix size of %d x %d and %d x %d\n",aelapsedTime,dim_param,dim_param,numARows,numAColumns,numBRows,numBColumns);

  //ali  wbTime_stop(Compute, "Performing CUDA computation");

  // wbLog(TRACE, "The block dimensions are ", dim_param, " x ", dim_param);
  //ali  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here

  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);
  
  //ali  wbTime_stop(Copy, "Copying output memory to the CPU");
  wbSolution(args, hostC, numCRows, numCColumns);
} /* end of block size sweep */
//ali  wbTime_start(GPU, "Freeing GPU Memory");

//@@ Free the GPU memory here

  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
 
//ali  wbTime_stop(GPU, "Freeing GPU Memory");
//ali  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
