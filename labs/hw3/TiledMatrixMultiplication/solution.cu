#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define TILE_WIDTH 16

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use tiling with shared memory for arbitrary size

  __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
  __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];
  
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  float Cvalue = 0;

  // A (m x k) * B (k x n) = C (m x n)
  // # rows in C = # rows in A
  // # columns in C = # columns in B

  for (int c = 0; c < (numAColumns - 1)/TILE_WIDTH + 1; ++c) {

    if (Row < numARows && (c * TILE_WIDTH + tx) < numAColumns)
      ds_A[ty][tx] = A[Row * numAColumns + (c * TILE_WIDTH + tx)];
    else
      ds_A[ty][tx] = 0.0;

    if ((c * TILE_WIDTH + ty) < numBRows && Col < numBColumns)
      ds_B[ty][tx] = B[(c * TILE_WIDTH + ty) * numBColumns + Col];
    else
      ds_B[ty][tx] = 0.0;

    __syncthreads();

    // if (Row < numCRows && Col < numCColumns) {
      for (int i = 0; i < TILE_WIDTH; ++i) {
        Cvalue += ds_A[ty][i] * ds_B[i][tx];
      }
    // }

    __syncthreads();
  }

  if (Row < numCRows && Col < numCColumns) {
    C[Row * numCColumns + Col] = Cvalue;
  }
  
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA; // A matrix on device
  float *deviceB; // B matrix on device
  float *deviceC; // C matrix on device
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C(you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
                            
  //@@ Set numCRows and numCColumns
  numCRows    = numARows;   // set to correct value
  numCColumns = numBColumns;   // set to correct value
  
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));
  
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
  wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);
  
  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here

  hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
  hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
  hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here

  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  // note that TILE_WIDTH is set to 16 on line number 13. 
  dim3 myBlock(TILE_WIDTH, TILE_WIDTH, 1);
  dim3 myGrid(ceil(numCColumns / TILE_WIDTH), ceil(numCRows / TILE_WIDTH), 1);
  // dim3 myGrid((numCColumns - 1)/TILE_WIDTH + 1, (numCRows - 1)/TILE_WIDTH + 1, 1);
  
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here

  matrixMultiplyShared<<<myGrid, myBlock>>>(deviceA, deviceB, deviceC, 
                                      numARows, numAColumns,
                                      numBRows, numBColumns, 
                                      numCRows, numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here

  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here

  free(deviceA);
  free(deviceB);
  free(deviceC);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
