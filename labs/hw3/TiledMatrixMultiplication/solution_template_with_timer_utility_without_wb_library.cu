#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define TILE_WIDTH 4

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use tiling with shared memory for arbitrary size

  __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
  __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];
  
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  float Cvalue = 0.0;  

  // A (m x k) * B (k x n) = C (m x n)
  // # rows in C = # rows in A
  // # columns in C = # columns in B

  for (int c = 0; c < ((numAColumns - 1)/TILE_WIDTH + 1); ++c) {

    if (Row < numARows && (c * TILE_WIDTH + tx) < numAColumns)
      ds_A[ty][tx] = A[Row * numAColumns + (c * TILE_WIDTH + tx)];
    else
      ds_A[ty][tx] = 0.0;

    if ((c * TILE_WIDTH + ty) < numBRows && Col < numBColumns)
      ds_B[ty][tx] = B[(c * TILE_WIDTH + ty) * numBColumns + Col];
    else
      ds_B[ty][tx] = 0.0;

    __syncthreads();

    if (Row < numCRows && Col < numCColumns) {
      for (int i = 0; i < TILE_WIDTH; ++i) {
        Cvalue += ds_A[ty][i] * ds_B[i][tx];
      }
    }

    __syncthreads();
  }

  if (Row < numCRows && Col < numCColumns) {
    C[Row * numCColumns + Col] = Cvalue;
  }

}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA; // A matrix on device
  float *deviceB; // B matrix on device
  float *deviceC; // C matrix on device
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)


  int dim_param;

hipEvent_t astartEvent, astopEvent;
float aelapsedTime;
hipEventCreate(&astartEvent);
hipEventCreate(&astopEvent);

  args = wbArg_read(argc, argv);

 //ali  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows    = numARows;   // set to correct value
  numCColumns = numBColumns;   // set to correct value
  
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));
  
 //ali  wbTime_stop(Generic, "Importing data and creating memory on host");

//ali  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
//ali  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
//ali  wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);
  
 //ali  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here for A, B and C
  
  hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
  hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
  hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));
  
 //ali wbTime_stop(GPU, "Allocating GPU memory.");

 //ali wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here for A and B
  
  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);
  
 //ali wbTime_stop(GPU, "Copying input memory to the GPU.");

  
  
  // ali: this loop will sweep blocks 4x4, 8x8, 16x16 and 32x32
  dim_param=4;
  for(dim_param=4; dim_param<33; dim_param=dim_param*2) {
    dim3 DimBlock(dim_param,dim_param,1);
    //@@ Initialize the grid dimensions here
    // use dim3 structure for setting grid dimensions
	dim3 DimGrid(ceil(numCColumns / dim_param), ceil(numCRows / dim_param));  // need to fill in DimGrid() call
    //ali  wbTime_start(Compute, "Performing CUDA computation");
    hipEventRecord(astartEvent, 0);
   //@@ Launch the GPU Kernel here
  
  matrixMultiplyShared<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, 
                                      numARows, numAColumns,
                                      numBRows, numBColumns, 
                                      numCRows, numCColumns);

  hipDeviceSynchronize();
  //hipDeviceSynchronize();

  hipEventRecord(astopEvent, 0);
  hipEventSynchronize(astopEvent);
  hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
  printf("Total execution time (ms) %f for block size %d x %d matrix size of %d x %d and %d x %d\n",aelapsedTime,dim_param,dim_param,numARows,numAColumns,numBRows,numBColumns);

  //ali  wbTime_stop(Compute, "Performing CUDA computation");

  // wbLog(TRACE, "The block dimensions are ", dim_param, " x ", dim_param);
  //ali  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  
  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);

  //ali  wbTime_stop(Copy, "Copying output memory to the CPU");
  wbSolution(args, hostC, numCRows, numCColumns);
} /* end of block size sweep */
//ali  wbTime_start(GPU, "Freeing GPU Memory");

//@@ Free the GPU memory here

  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
 
//ali  wbTime_stop(GPU, "Freeing GPU Memory");
//ali  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
