#include "hip/hip_runtime.h"
/* 
Histogram generation algorithm for an input array of integers within a given range. 
Each integer will map into a single bin, so the values will range from 0 to (NUM_BINS - 1). 
The histogram bins will use unsigned 32-bit counters that must be saturated at 127, 
meaning all bins with values larger than 127 need to be clipped to 127.  
This clipping operation is a key step during histogram equalization process. 
The input length can be assumed to be less than 2^32. NUM_BINS is fixed at 4096.
*/

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>

// version 0
// global memory only interleaved version
// include comments describing your approach
__global__ void histogram_global_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

	// insert your code here
	int i = threadIdx.x + blockIdx.x * blockDim.x; // index
	int stride = blockDim.x * gridDim.x; // total number of threads

	while (i < num_elements) {
		int pos = input[i]; // bin position
		if (pos >= 0 && pos < num_bins) // boundary condition check
			atomicAdd(&bins[pos], 1); // atomically increment appropriate bin
		i += stride;
	}
}


// version 1
// shared memory privatized version
// include comments describing your approach
__global__ void histogram_shared_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

	// insert your code here
	__shared__ unsigned int bins_private[4096]; // privatized bins
	int i = threadIdx.x + blockIdx.x * blockDim.x; // index
	int stride = blockDim.x * gridDim.x; // total number of threads

	// initialize privatized bins to 0
	if (threadIdx.x < 4096) bins_private[threadIdx.x] = 0;
	__syncthreads();

	// build local histogram
	while (i < num_elements) {
		int pos = input[i]; // bin position
		if (pos >= 0 && pos < 4096) // boundary condition check
			atomicAdd(&bins_private[pos], 1); // atomically increment appropriate privatized bin
		i += stride;
	}
	__syncthreads();

	// build global histogram
	// number of bins > block size -- need multiple bins per thread
	for (int j = 0; j < num_bins; j += blockDim.x) {
		atomicAdd(&bins[threadIdx.x + j], bins_private[threadIdx.x + j]);
	}
}


// version 2
// your method of optimization using shared memory 
// include DETAILED comments describing your approach
__global__ void histogram_shared_accumulate_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

	// insert your code here
	int i = threadIdx.x + blockIdx.x * blockDim.x; // index
	int stride = blockDim.x * gridDim.x; // total number of threads
	// __shared__ unsigned int bins_private[4096]; // privatized bins

	thrust::device_vector<unsigned int> input_sort(input); // copy input data
	thrust::sort(input_sort.begin(), input_sort.end()); // sort input 

	thrust::device_vector<unsigned int> histo_values;
	thrust::device_vector<unsigned int> histo_counts;
	histo_values.resize(4096);
  	histo_counts.resize(4096);

	thrust::reduce_by_key(input_sort.begin(), input_sort.end(), thrust::constant_iterator<int>(1), histo_values.begin(), histo_counts.begin());

	for (int j = 0; j < num_bins; j += blockDim.x) {
		atomicAdd(&bins[threadIdx.x + j], histo_counts[threadIdx.x + j]);
	}
	
	// sorting based approach
	// reduce by key
	// compression before reduction

}

// clipping function
// resets bins that have value larger than 127 to 127. 
// that is if bin[i]>127 then bin[i]=127
__global__ void convert_kernel(unsigned int *bins, unsigned int num_bins) {

	// insert your code here
	int i = threadIdx.x + blockIdx.x * blockDim.x; // index
	if (bins[i] > 127) bins[i] = 127;
}
